#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cmath>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <limits.h>
#include <math.h>
#include <hipcub/hipcub.hpp>

#include <sstream>
#include <vector>
#include <fstream>
#include <string>
#include <cstring>
#include <cstdlib>
#include <ctime>

#include "para.cuh"
#include "F1.cuh"
#include "F2.cuh"
#include "F4.cuh"

__host__ __device__ double F5(double *x, double *Ovector, double *anotherz, int *Pvector, double *r25, double *r50, double *r100)
{
     int    i;
    double result=0.0;
    //double anotherz[1000];
    double anotherz1[100];
    int s[7];
    s[0] = 50;
    s[1] = 25;
    s[2] = 25;
    s[3] = 100;
    s[4] = 50;
    s[5] = 25;
    s[6] = 25;

    double w[7];
    w[0] = 0.1807559981875739;
    w[1] = 9081.137957372908;
    w[2] = 24.27180711217444;
    w[3] = 1.86308808032975e-06;
    w[4] = 17698.08075760589;
    w[5] = 0.0002815181918094626;
    w[6] = 0.01525403796219806;

    for(i = 0; i < N; i++) {
        anotherz[i] = x[i] - Ovector[i];
    }

    int* c = new int[1];
    c[0] = 0;
    for (i = 0; i < 7; i++)
    {
        rotateVector(i, c, anotherz1, s, anotherz, Pvector, r25, r50, r100);
        result += w[i] * rastrigin(anotherz1, s[i]);
        //delete []anotherz1;
    }
    //double* z = new double[dimension-c];
    double z[700];
    //double* z = (double*) malloc(sizeof(double)*(N-c));
    //double* z = new double [N-c[0]];
    for (i = 300; i < N; i++)
        z[i-300] = anotherz[Pvector[i]];

    result += rastrigin(z, 700);
    delete[] c;
    //return(result);
    return result;
}

